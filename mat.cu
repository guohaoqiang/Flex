#include "hip/hip_runtime.h"
#include "mat.cuh"
#include <bit>
#include <ranges>

__constant__ Mat_POD mat_dev;

Mat::Mat(DataLoader& input, int tileh,int tilew)
         :dl(input),rowPtr(input.rowPtr),colIdx(input.col),vals(input.vals),voMp(input.vo_mp){
            m = input.n;
            n = m;
            k = input.dim;
            nnz = input.nnz;
			tm = tileh;
            tn = tilew;
			tileRowPtr.push_back(0);
			segPtr.push_back(0);
			tileNnz.push_back(0);
			newVals.resize(input.nnz);
			pos = 0;
            bitMap_bytes = 0; 
            voMp_bytes = 0; 
            nnz_limit = NNZ_LIMIT;
            atomic_op = 0;
}
void Mat::launch_prep(){
    dl.gpuC_zero();
    mat_b_dev = dl.gpuX;
    if (dl.vertex_order_abbr == "OVO"){
       shadow_b_dev = dl.gpuX; 
    }
    mat_c_dev = dl.gpuC;
    Mat_POD for_dev(*this);
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(mat_dev), &for_dev, sizeof(for_dev), 0, hipMemcpyHostToDevice));
}
void Mat::transfer(){
#   define CMALC(var)                                   \
     var##_bytes = var.size() * sizeof( var[0] );        \
     CHECK_CUDA(hipMalloc( &var##_dev, var##_bytes )) ;

     CMALC( tileNnz ); CMALC( tileColIdx ); CMALC( vals );
     CMALC( tileRowPtr ); CMALC( nnzTile ); CMALC( rcOffset );
#ifndef COL_MAJ_TILE
CMALC( bitMap );
#endif
#ifdef VO_RECOVER
CMALC( voMp );
#endif
#   undef CMALC

    // transfer data to device
    hipMemcpy(tileNnz_dev, tileNnz.data(), tileNnz.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tileColIdx_dev, tileColIdx.data(), tileColIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vals_dev, newVals.data(), newVals.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tileRowPtr_dev, tileRowPtr.data(), tileRowPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nnzTile_dev, nnzTile.data(), nnzTile.size()*sizeof(int), hipMemcpyHostToDevice);
#ifndef COL_MAJ_TILE
    hipMemcpy(bitMap_dev, bitMap.data(), bitMap.size()*sizeof(int), hipMemcpyHostToDevice);
#endif
    hipMemcpy(rcOffset_dev, rcOffset.data(), rcOffset.size()*sizeof(int), hipMemcpyHostToDevice);
#ifdef VO_RECOVER
    hipMemcpy(voMp_dev, voMp.data(), voMp.size()*sizeof(int), hipMemcpyHostToDevice);
    if (dl.vertex_order_abbr != "OVO"){
        CHECK_CUDA(hipMalloc( &shadow_b_dev,  m*k*sizeof(float))) ;
        CHECK_CUDA(hipMemset( shadow_b_dev,  0, m*k*sizeof(float))) ;
    }
#endif
}
void Mat::transfer2(){
#   define CMALC(var)                                   \
     var##_bytes = var.size() * sizeof( var[0] );        \
     CHECK_CUDA(hipMalloc( &var##_dev, var##_bytes )) ;

     CMALC( segPtr ); CMALC( segNzRCIdx ); CMALC( segNzRowIdx ); CMALC( segNzColIdx ); 
     CMALC( vals ); CMALC( voMp ); CMALC( segVoMap );
#   undef CMALC

    // transfer data to device
    hipMemcpy(segNzRCIdx_dev, segNzRCIdx.data(), segNzRCIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segNzRowIdx_dev, segNzRowIdx.data(), segNzRowIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segNzColIdx_dev, segNzColIdx.data(), segNzColIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vals_dev, newVals.data(), newVals.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(segPtr_dev, segPtr.data(), segPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segVoMap_dev, segVoMap.data(), segVoMap.size()*sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(voMp_dev, voMp.data(), voMp.size()*sizeof(int), hipMemcpyHostToDevice);
    if (dl.vertex_order_abbr != "OVO"){
        CHECK_CUDA(hipMalloc( &shadow_b_dev,  m*k*sizeof(float))) ;
        CHECK_CUDA(hipMemset( shadow_b_dev,  0, m*k*sizeof(float))) ;
    }
}
void Mat::dataVolume_est2(){
    est_fp = int64_t(nnz)*k;
    // shadow_b_bytes is identical to gpuX_bytes when perform v9
    // so dl.gpuX_bytes can be seen shadow_b_bytes when v9
    est_ld_bytes = int64_t(segNzRowIdx_bytes) + 
                    segNzColIdx_bytes + 
                    vals_bytes + 
                    dl.gpuX_bytes +
                    segPtr_bytes + 
                    segVoMap_bytes;
    est_st_bytes = dl.gpuC_bytes;
}
void Mat::dataVolume_est(){
    est_fp = int64_t(nnz)*k;
    // shadow_b_bytes is identical to gpuX_bytes when perform v9
    // so dl.gpuX_bytes can be seen shadow_b_bytes when v9
    est_ld_bytes = int64_t(tileNnz_bytes) + 
                    tileColIdx_bytes + 
                    vals_bytes + 
                    dl.gpuX_bytes +
                    tileRowPtr_bytes + 
                    nnzTile_bytes + 
                    bitMap_bytes + 
                    rcOffset_bytes +
                    voMp_bytes;
    est_st_bytes = dl.gpuC_bytes;
}

void Mat::csr2tile(){
	
	int tileRows = (m+tm-1)/tm;
	for (int i=0; i<tileRows; ++i){
		//csr2flex_Rmajor(i);
		//csr2flex_Cmajor(i);
		//csr2regular(i);
        csr2seg_Cmajor(i);
	} 
    n_segs = segPtr.size()-1;
}
void Mat::print3(int l){
    if ( false ){
        printf("\nSegPtr: \n");
        for (int i=0; i<l?l:segPtr.size(); ++i){
            printf("(%d:%d)  ",i,segPtr[i]);
        }
        printf("\nSegRowNzIdx: %d\n",(int)segNzRowIdx.size());
        for (int i=0; i<l?l:segNzRowIdx.size(); ++i){
            printf("%d  ",segNzRowIdx[i]);
        }
        printf("\nSegColNzIdx: %d\n",(int)segNzColIdx.size());
        for (int i=0; i<l?l:segNzColIdx.size(); ++i){
            printf("%d  ",segNzColIdx[i]);
        }
    }
    printf("\nSegVoMap: %d\n",(int)segVoMap.size());
    for (int i=0; i<(l?l:segVoMap.size()); ++i){
        printf("%d->%d  ",i,segVoMap[i]&0x7fffffff);
    }
    printf("\n");
}

void Mat::csr2seg_Cmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

	// keep track of the cols in each row
	std::vector<int> cOffset(tm, 0);
	
    int dif = 0.1*nnz_limit; 
    int nnzInSeg = 0;
    int nnz_cur_panel = rowPtr[rowEnd] - rowPtr[rowStart];    
    vector<int> atom(tm, 0);

    map<int,int> occ_cols;
    for ( auto c: views::iota(rowPtr[rowStart],rowPtr[rowEnd]) )
      occ_cols[colIdx[c]]++;
    const auto last_col = occ_cols.rbegin()->first;

    // collect segs in the panel
    for ( auto [j,ncol]: occ_cols ) {
        
        for ( int i=rowStart; i<rowEnd; ++i ){
            // absolute position of the nze in csr, idx = base + offset
            int c = rowPtr[i] + cOffset[i-rowStart];
            if ( colIdx[c]==j && c<rowPtr[i+1] ){
                // nze values
                segNzRowIdx.push_back(i-rowStart);
                segNzColIdx.push_back(j);
                segNzRCIdx.push_back(i-rowStart);
                segNzRCIdx.push_back(j);
                newVals[pos++] = vals[c];
                cOffset[i-rowStart]++;
                atom[i-rowStart]++;
                nnzInSeg++;
            }
        }
        if ( (j==last_col && nnzInSeg) || (nnz_limit - nnzInSeg)<=dif || nnzInSeg>nnz_limit ){
         
            segPtr.push_back(segPtr.back()+nnzInSeg);
            nnzInSeg = 0;
            
            for (int i=rowStart; i<rowStart+tm; ++i){
                if ( i<rowEnd ){
                    if ( atom[i-rowStart]>=0 && atom[i-rowStart]<(rowPtr[i+1]-rowPtr[i]) ){
                        // if the #nz in a specific row of a seg 
                        // is less than that of the whole row,
                        // the row requires "atomic add".
                        // use MSB to mark it.
                        segVoMap.push_back( voMp[i] | (1<<31) );
                    }else{ 
                        segVoMap.push_back( voMp[i] );
                    }
                }else{
                    // for the last panel, the rows may be less than tm 
                    segVoMap.push_back(1<<(bit_width((uint)m)+1));
                }
                
                atom[ i-rowStart ] = 0;
            }
        }
    }
}

void
Mat::stats_collect2(FILE *stream)
{
  //const uint seg_m = ( m + tm - 1 ) / tm;
  //const uint seg_m_floor = m / tm;
  
  const uint seg_nnz_lim = tm * n;
  assert( seg_nnz_lim == tm * uint64_t(n) ); // Overflow check.
  const uint seg_lg_nnz_lim = bit_width(seg_nnz_lim);
  uint seg_lg_nnz_max = 0, seg_lg_nnz_min = seg_lg_nnz_lim;
  seg_lg_nnz_histo.resize(seg_lg_nnz_lim+1);

  const uint n_segs = segPtr.size()-1;

  n_col_sum = 0;
  int sp_seg1 = 0;
  int sp_seg2 = 0;
  int sp_seg3 = 0;
  int sp_seg4 = 0;
  for ( uint seg_idx = 0; seg_idx < n_segs; seg_idx++ )
    {
      const uint nnz_seg = segPtr[seg_idx+1] - segPtr[seg_idx];
      if ( nnz_seg<=NNZ_LIMIT/4 ) sp_seg1++;
      else if ( nnz_seg<=NNZ_LIMIT/2 ) sp_seg2++;
      else if ( nnz_seg<=NNZ_LIMIT ) sp_seg3++;
      else if ( nnz_seg>NNZ_LIMIT ) sp_seg4++;
      
      const uint lg_nnz = bit_width(nnz_seg);
      set_max( seg_lg_nnz_max, lg_nnz );
      set_min( seg_lg_nnz_min, lg_nnz );
      seg_lg_nnz_histo[lg_nnz]++;
      
      unordered_set<int> colset; 
      for (int i=segPtr[seg_idx]; i<segPtr[seg_idx+1]; ++i){
        colset.insert(segNzColIdx[i]);
      }
      n_col_sum += colset.size();
    }

  for ( uint32_t v: segVoMap ) if ( v & (1u<<31) ) atomic_op++;

  if ( !stream ) return;

  fprintf(stream, "Ordering %s.  Segments %d × *\n",
          dl.vertex_order_abbr.c_str(), tm);

  fprintf(stream, "Histogram of lg non-zeros per tile-segment.\n");

  pTable lg_nnz_tab(stream);
  for ( auto i: views::iota(seg_lg_nnz_min,seg_lg_nnz_max+1) )
    {
      pTable_Row _(lg_nnz_tab);
      lg_nnz_tab.entry("Lg", "%2d", i);
      lg_nnz_tab.entry("Seg", "%7d", seg_lg_nnz_histo[i] );
      lg_nnz_tab.entry
        ("Pct", "%6.2f", seg_lg_nnz_histo[i] * 100.0 / n_segs );
    }

  fprintf(stream,"Arrays m=%d, n=%d, k=%d. tile-seg %d × *.\n"
          "              n_segs=%u, nnz=%d  Avg deg=%.1f\n",
         m, n, k, tm, n_segs, nnz, double(nnz)/m);

  fprintf(stream,"nnz / seg: %.3f     Load / B elt  %.3f\n",
         double(nnz) / n_segs,
         n_col_sum / double(nnz) );
  fprintf(stream,"nnz in segs:  (0,%d]: %d  %.3f     (%d,%d]: %d  %.3f     (%d,%d]: %d  %.3f      (%d,%d]: %d  %.3f\n",
         NNZ_LIMIT/4, sp_seg1, double(sp_seg1) / n_segs,
         NNZ_LIMIT/4, NNZ_LIMIT/2, sp_seg2, double(sp_seg2) / n_segs,
         NNZ_LIMIT/2, NNZ_LIMIT, sp_seg3, double(sp_seg3) / n_segs,
         NNZ_LIMIT, NNZ_LIMIT+tm, sp_seg4, double(sp_seg4) / n_segs );
  fprintf(stream,"\n");
}

// convert a row of tiles to FlexSpTiles
void Mat::csr2flex_Rmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

        const int n_tiles_limit = ( n + tn - 1 ) / tn;

	// keep track of the cols in each row
	std::vector<int> cIdx(tm, -1); 
	std::vector<int> cOffset(tm, 0);
	// get the left bound
	// iterate over rows to get the smallest col idx
	unsigned int left = n;
	for (int i=rowStart; i<rowEnd; ++i){
		// here, we assume there is no empty row
		left = min((int)left, (int)colIdx[rowPtr[i]]);
		cIdx[i-rowStart] = colIdx[rowPtr[i]];
	}

	// right bound (exclusive)
	unsigned int right = min((int)left + tn, n);
	int nnzInRows = 0;
    int tiles_in_cur_row = 0;

    //int tileStart = rowPtr[ridx];
    while (pos<rowPtr[rowEnd]){
		int nnzInTile = 0;
        tiles_in_cur_row++;
        assert( tiles_in_cur_row <= n_tiles_limit );
		// collect tiles in the tile-row
        int bit_map = 0;
		for (int i=rowStart; i<rowEnd; ++i){
			// absolute position of the nze in csr, idx = base + offset
			int c = rowPtr[i] + cOffset[i-rowStart];
			//  #nze in the i-th row
			
			// c check is necessary because it constraines nze within the i-th row
                        while ( c<rowPtr[i+1] && colIdx[c]<right ){
                //char rc = 0;
                int rc16 = 0;

				// currently, it is not 4-bit
				int temp_rowOffset = i-rowStart;
                //rc |= (temp_rowOffset<<4);
                rc16 |= (temp_rowOffset<<16);

				// real col idx
				int temp_tileColIdx = cIdx[i-rowStart];
                                assert( temp_tileColIdx >= left );
                //rc |= (temp_tileColIdx-left);
                rc16 |= (temp_tileColIdx-left);
			    bit_map |= 1<<(temp_tileColIdx-left);	
                
                // nze values
				newVals[pos] = vals[c];
                rcOffset.push_back(rc16);

				cIdx[i-rowStart] = colIdx[++c];
				pos++;
				cOffset[i-rowStart]++;
				nnzInTile++;
				nnzInRows++;
			}
		}
        
        // ---------- v4 -------
        //tileStart = tileNnz.back()+nnzInTile; 
        // mark the last tile in current row-tile
        //if (pos>=rowPtr[rowEnd]){
        //    nnzInTile |= (1<<31);
        //}
        nnzTile.push_back(nnzInTile); 
        bitMap.push_back(bit_map); 
        // ---------------------
		
		tileNnz.push_back(tileNnz.back()+nnzInTile);
        tileColIdx.push_back(left);
        // update left and right bound for next tile
		left = n;
		for (int i=rowStart; i<rowEnd; ++i){
			// check whether the column goes to the next row
			int rnnz = rowPtr[i+1]-rowPtr[i];
			if (cOffset[i-rowStart]<rnnz){
				left = min((int)left, (int)cIdx[i-rowStart]);
			}
		}
		right = min((int)left + tn, n);

	}
	tileRowPtr.push_back(tileRowPtr.back()+tiles_in_cur_row);
}

// convert a row of tiles to FlexSpTiles
void Mat::csr2flex_Cmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

    const int n_tiles_limit = ( n + tn - 1 ) / tn;

	// keep track of the cols in each row
	std::vector<int> cIdx(tm, -1); 
	std::vector<int> cOffset(tm, 0);
	// get the left bound
	// iterate over rows to get the smallest col idx
	unsigned int left = n;
	for (int i=rowStart; i<rowEnd; ++i){
		// here, we assume there is no empty row
		left = min((int)left, (int)colIdx[rowPtr[i]]);
		cIdx[i-rowStart] = colIdx[rowPtr[i]];
	}

	// right bound (exclusive)
	unsigned int right = min((int)left + tn, n);
	int nnzInRows = 0;
    int tiles_in_cur_row = 0;

    //int tileStart = rowPtr[ridx];
    while (pos<rowPtr[rowEnd]){
		int nnzInTile = 0;
        tiles_in_cur_row++;
        assert( tiles_in_cur_row <= n_tiles_limit );
		// collect tiles in the tile-row
        int bit_map = 0;
        for (int i_tn=0; i_tn<tn; ++i_tn){
            
		    for (int i=rowStart; i<rowEnd; ++i){
                // absolute position of the nze in csr, idx = base + offset
                int c = rowPtr[i] + cOffset[i-rowStart];
                if ( colIdx[c]==left+i_tn && c<rowPtr[i+1] && colIdx[c]<right ){
                    int rc16 = 0;

                    // currently, it is not 4-bit
                    int temp_rowOffset = i-rowStart;
                    //rc |= (temp_rowOffset<<4);
                    rc16 |= (temp_rowOffset<<16);

                    // real col idx
                    int temp_tileColIdx = cIdx[i-rowStart];
                    assert( temp_tileColIdx >= left );
                    //rc |= (temp_tileColIdx-left);
                    rc16 |= (temp_tileColIdx-left);
			        bit_map |= 1<<(temp_tileColIdx-left);	
                    
                    // nze values
                    newVals[pos] = vals[c];
                    rcOffset.push_back(rc16);

                    cIdx[i-rowStart] = colIdx[++c];
                    pos++;
                    cOffset[i-rowStart]++;
                    nnzInTile++;
                    nnzInRows++; 
                }
            }
        }
        nnzTile.push_back(nnzInTile); 	
        bitMap.push_back(bit_map); 
		
        tileNnz.push_back(tileNnz.back()+nnzInTile);
        tileColIdx.push_back(left);
        // update left and right bound for next tile
		left = n;
		for (int i=rowStart; i<rowEnd; ++i){
			// check whether the column goes to the next row
			int rnnz = rowPtr[i+1]-rowPtr[i];
			if (cOffset[i-rowStart]<rnnz){
				left = min((int)left, (int)cIdx[i-rowStart]);
			}
		}
		right = min((int)left + tn, n);
	}
	tileRowPtr.push_back(tileRowPtr.back()+tiles_in_cur_row);
}

void
Mat::stats_collect(FILE *stream)
{
  const uint tmn = tm * tn;

  const uint tile_m = ( m + tm - 1 ) / tm;
  const uint tile_m_floor = m / tm;
  const uint tile_n = ( n + tn - 1 ) / tn;
  tile_p_row_histo.resize(tile_n+1);
  uint max_n_tiles = 0;

  const uint panel_nnz_lim = tm * n;
  assert( panel_nnz_lim == tm * uint64_t(n) ); // Overflow check.
  const uint panel_lg_nnz_lim = bit_width(panel_nnz_lim);
  uint panel_lg_nnz_max = 0, panel_lg_nnz_min = panel_lg_nnz_lim;
  panel_lg_nnz_histo.resize(panel_lg_nnz_lim+1);

  for ( uint tile_r = 0;  tile_r < tile_m;  tile_r++ )
    {
      const uint n_tiles = tileRowPtr[tile_r+1] - tileRowPtr[tile_r];
      assert( n_tiles <= tile_n );
      set_max( max_n_tiles, n_tiles );
      tile_p_row_histo[n_tiles]++;
      if ( tile_r >= tile_m_floor ) continue;
      const auto tile_start = tileRowPtr[tile_r];
      const auto tile_stop = tileRowPtr[tile_r+1];
      const auto tidx_start = tileNnz[tile_start];
      const auto tidx_stop = tileNnz[tile_stop];
      if (tidx_stop <= tidx_start){
        printf("tm = %d, tn = %d, tidx_stop = %d, tidx_satrt = %d\n",tm, tn, tidx_stop, tidx_start);
      }
      assert( tidx_stop > tidx_start );
      const uint nnz_panel = tidx_stop - tidx_start;
      const uint lg_nnz = bit_width(nnz_panel);
      assert( lg_nnz < panel_lg_nnz_histo.size() );
      assert( lg_nnz );
      set_max( panel_lg_nnz_max, lg_nnz );
      set_min( panel_lg_nnz_min, lg_nnz );
      panel_lg_nnz_histo[lg_nnz]++;
    }
  tile_p_row_histo.resize(max_n_tiles+1);
  const uint n_tiles = nnzTile.size();

  tile_nnz_histo.resize(tmn+1);
  n_col_sum = 0;
  uint max_t_nnz = 0;
  for ( uint t_idx = 0; t_idx < n_tiles; t_idx++ )
    {
      const auto nnz = nnzTile[t_idx];
      tile_nnz_histo[nnz]++;
      set_max( max_t_nnz, nnz );
      const uint n_col = popcount(uint(bitMap[t_idx]));
      n_col_sum += n_col;
    }
  tile_nnz_histo.resize(max_t_nnz+1);

  vector<int> tiles_bucket(6,0);
  int total = 0;
  int remain = 0;
  for (int i=0; i<tile_p_row_histo.size(); ++i){
    int counts = tile_p_row_histo[i]; 
    if (counts>=1 && counts<8){
        tiles_bucket[0] += counts;
    }else if (counts>=8 && counts<16){
        tiles_bucket[1] += counts;
    }else if (counts>=16 && counts<32){
        tiles_bucket[2] += counts;
    }else if (counts>=32 && counts<64){
        tiles_bucket[3] += counts;
    }else if (counts>=64 && counts<128){
        tiles_bucket[4] += counts;
    }else if (counts>=128){
        tiles_bucket[5] += counts;
    }else{
        remain += counts;
    }
    total += counts;
  }
  if ( !stream ) return;

  fprintf(stream, "Ordering %s.  Tile %d × %d\n",
          dl.vertex_order_abbr.c_str(), tm, tn);

  fprintf(stream, "Histogram of lg non-zeros per panel (tile row).\n");

  pTable lg_nnz_tab(stream);
  for ( auto i: views::iota(panel_lg_nnz_min,panel_lg_nnz_max+1) )
    {
      pTable_Row _(lg_nnz_tab);
      lg_nnz_tab.entry("Lg", "%2d", i);
      lg_nnz_tab.entry("Panels", "%7d", panel_lg_nnz_histo[i] );
      lg_nnz_tab.entry
        ("Pct", "%6.2f", panel_lg_nnz_histo[i] * 100.0 / tile_m_floor );
    }

  fprintf(stream,"[1,8): %f%%   ", tiles_bucket[0]*100.0/tile_m); 
  fprintf(stream,"[8,16): %f%%    ", tiles_bucket[1]*100.0/tile_m); 
  fprintf(stream,"[16,32): %f%%   ", tiles_bucket[2]*100.0/tile_m); 
  fprintf(stream,"[32,64): %f%%   ", tiles_bucket[3]*100.0/tile_m); 
  fprintf(stream,"[64,128): %f%%  ", tiles_bucket[4]*100.0/tile_m); 
  fprintf(stream,"[128, +OO): %f%%\n", tiles_bucket[5]*100.0/tile_m); 

  fprintf(stream,"Arrays m=%d, n=%d, k=%d. Tile %d × %d.   nnz=%d  Avg deg=%.1f\n",
         m, n, k, tm, tn, nnz, double(nnz)/m);

  fprintf(stream,"nnz / tile: %.3f  Load / B elt  %.3f\n",
         double(nnz) / n_tiles,
         n_col_sum / double(nnz) );
  int n_t_hist_pr = 0;
  fprintf(stream,"Tile nnz histogram: (n_tiles %d)\n",n_tiles);
  for ( uint i=0; i<tile_nnz_histo.size(); i++ )
    if ( auto tnnz = tile_nnz_histo[i]; tnnz )
      {
        if ( n_t_hist_pr++ > 6 ) break;
        fprintf(stream,"%3d %5.2f%%, ", i, tnnz * 100.0 / n_tiles);
      }
  fprintf(stream,"\n");
}


void Mat::print2(){
#ifdef DEBUG
    for (int i=0; i<tileRowPtr.size(); ++i)
		std::cout<<tileRowPtr[i]<<" ";
	std::cout<<std::endl;
    
	/*	
    for (int i=0; i<tileLeftColIdx.size(); ++i)
		std::cout<<tileLeftColIdx[i]<<" ";
	std::cout<<std::endl;
    std::cout<<"------- tile elements: -------"<<std::endl;
	std::cout<<std::endl;
	for (int i=0; i<tileColIdx.size(); ++i)
		std::cout<<tileColIdx[i]<<" ";
    std::cout<<std::endl<<"rc:"<<std::endl;
	for (int i=0; i<rc_Offset.size(); ++i)
		std::cout<<(int)rc_Offset[i]<<" ";
	std::cout<<std::endl;
	for (int i=0; i<newVals.size(); ++i)
		std::cout<<newVals[i]<<" ";
	std::cout<<std::endl;
    */
#endif
    std::cout<<std::endl<<"nnzTile:"<<std::endl;
    for (int i=0; i<nnzTile.size(); ++i){
    //for (int i=0; i<20; ++i){
        std::cout<<nnzTile[i]<<" ";
    }
#ifndef COL_MAJ_TILE
    std::cout<<std::endl<<"bitMap:"<<std::endl;
    //for (int i=0; i<bitMap.size(); ++i){
    for (int i=0; i<20; ++i){
        std::cout<<bitMap[i]<<" ";
    }
#endif
    std::cout<<std::endl<<"rc:"<<std::endl;
	for (int i=0; i<rcOffset.size(); ++i){
	//for (int i=0; i<20; ++i){
		int r = rcOffset[i]>>16;
		int c = rcOffset[i] & 0x0000FFFF;
        std::cout<<"{"<<r<<","<<c<<"}"<<" ";
    }
	
//    std::cout<<std::endl<<"vals:"<<std::endl;
//	for (int i=0; i<newVals.size(); ++i)
//		std::cout<<newVals[i]<<" ";
	std::cout<<std::endl;
	std::cout<<"Flex Tiles: "<<tileNnz.size()-1<<std::endl;
}
