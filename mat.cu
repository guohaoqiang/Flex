#include "hip/hip_runtime.h"
#include "mat.cuh"
#include <bit>
#include <ranges>

__constant__ Mat_POD mat_dev;

Mat::Mat(DataLoader& input, int tileh,int tilew)
         :dl(input),rowPtr(input.rowPtr),colIdx(input.col),vals(input.vals),voMp(input.vo_mp){
            m = input.n;
            n = m;
            k = input.dim;
            nnz = input.nnz;
			tm = tileh;
            tn = tilew;
            uni_nb = input.uni_nb;
			tileRowPtr.push_back(0);
			segPtr.push_back(0);
			tileNnz.push_back(0);
			newVals.resize(input.nnz);
			pos = 0;
            bitMap_bytes = 0; 
            voMp_bytes = 0; 
            nnz_limit = NNZ_LIMIT;
            atomic_op = 0;


            csr_rowPtr_dev = dl.rowPtr_dev;
            csr_col_dev = dl.col_dev;
            csr_vals_dev = dl.vals_dev;
            csr_mat_b_dev = dl.gpuX;
}
void Mat::launch_prep(){
    dl.gpuC_zero();
    mat_b_dev = dl.gpuX;
    if (dl.vertex_order_abbr == "OVO"){
       shadow_b_dev = dl.gpuX; 
    }
    mat_c_dev = dl.gpuC;
    Mat_POD for_dev(*this);
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(mat_dev), &for_dev, sizeof(for_dev), 0, hipMemcpyHostToDevice));
}
void Mat::transfer(){
#   define CMALC(var)                                   \
     var##_bytes = var.size() * sizeof( var[0] );        \
     CHECK_CUDA(hipMalloc( &var##_dev, var##_bytes )) ;

     CMALC( tileNnz ); CMALC( tileColIdx ); CMALC( vals );
     CMALC( tileRowPtr ); CMALC( nnzTile ); CMALC( rcOffset );
#ifndef COL_MAJ_TILE
CMALC( bitMap );
#endif
#ifdef VO_RECOVER
CMALC( voMp );
#endif
#   undef CMALC

    // transfer data to device
    hipMemcpy(tileNnz_dev, tileNnz.data(), tileNnz.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tileColIdx_dev, tileColIdx.data(), tileColIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vals_dev, newVals.data(), newVals.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tileRowPtr_dev, tileRowPtr.data(), tileRowPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nnzTile_dev, nnzTile.data(), nnzTile.size()*sizeof(int), hipMemcpyHostToDevice);
#ifndef COL_MAJ_TILE
    hipMemcpy(bitMap_dev, bitMap.data(), bitMap.size()*sizeof(int), hipMemcpyHostToDevice);
#endif
    hipMemcpy(rcOffset_dev, rcOffset.data(), rcOffset.size()*sizeof(int), hipMemcpyHostToDevice);
#ifdef VO_RECOVER
    hipMemcpy(voMp_dev, voMp.data(), voMp.size()*sizeof(int), hipMemcpyHostToDevice);
    if (dl.vertex_order_abbr != "OVO"){
        CHECK_CUDA(hipMalloc( &shadow_b_dev,  m*k*sizeof(float))) ;
        CHECK_CUDA(hipMemset( shadow_b_dev,  0, m*k*sizeof(float))) ;
    }
#endif
}
void Mat::transfer2(){
#   define CMALC(var)                                   \
     var##_bytes = var.size() * sizeof( var[0] );        \
     CHECK_CUDA(hipMalloc( &var##_dev, var##_bytes )) ;

     CMALC( segPtr ); CMALC( segNzRCIdx ); CMALC( segNzRowIdx ); CMALC( segNzColIdx ); 
     CMALC( vals ); CMALC( voMp ); CMALC( segVoMap ); 
     CMALC( grouped_tailSeg ); CMALC( next_seg );
     CMALC( seg_rowPtr ); CMALC( segNzCV );
#   undef CMALC

    // transfer data to device
    hipMemcpy(segNzRCIdx_dev, segNzRCIdx.data(), segNzRCIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segNzRowIdx_dev, segNzRowIdx.data(), segNzRowIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segNzColIdx_dev, segNzColIdx.data(), segNzColIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vals_dev, newVals.data(), newVals.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(segPtr_dev, segPtr.data(), segPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segVoMap_dev, segVoMap.data(), segVoMap.size()*sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(seg_rowPtr_dev, seg_rowPtr.data(), seg_rowPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(segNzCV_dev, segNzCV.data(), segNzCV.size()*sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(voMp_dev, voMp.data(), voMp.size()*sizeof(int), hipMemcpyHostToDevice);
    if (dl.vertex_order_abbr != "OVO"){
        CHECK_CUDA(hipMalloc( &shadow_b_dev,  m*k*sizeof(float))) ;
        CHECK_CUDA(hipMemset( shadow_b_dev,  0, m*k*sizeof(float))) ;
    }
    hipMemcpy(grouped_tailSeg_dev, grouped_tailSeg.data(), grouped_tailSeg.size()*sizeof(int), hipMemcpyHostToDevice);
}
void Mat::dataVolume_est2(){
  
/*********** compute B rows to be loaded for each sm **************************/
    int64_t validate_nnz = 0; 
    vector<unordered_set<int>> col_st(sms+1, unordered_set<int>());  
    unordered_map<int,int> c_sm;
    // the first SM buckets 
    for (int i=0; i<sms; ++i){
        
        if ( next_seg[ i ]<grouped_tailSeg[ i ] ){
            for (int j=seg_rowPtr[ next_seg[ i ]*(tm+1) ]; 
                    j<seg_rowPtr[ grouped_tailSeg[ i ]*(tm+1) -1]; ++j){
                validate_nnz++;
                col_st[i].insert( (int)segNzCV[j*2] );
            }
        }
        // collect long row break and draw the pie
        for ( auto &cc: col_st[i] ){
            c_sm[cc]++;
        }
    }
    
    // the last bucket, which is used for workload balance
    acc_col = 0;
    for (int i=next_seg[sms]; i<n_segs; ++i){
       unordered_set<int> last_tile_col;
       for (int ii=0; ii<tm; ++ii) 
        for (int j=seg_rowPtr[ i*(tm+1)+ii ]; 
                j<seg_rowPtr[ i*(tm+1)+ii+1 ]; ++j){
            validate_nnz++;
            col_st[sms].insert( (int)segNzCV[j*2] );
            last_tile_col.insert( (int)segNzCV[j*2] );
        }

       acc_col += last_tile_col.size();
        for ( auto &cc: col_st[sms] ){
            c_sm[cc]++;
        }
    }

    if (false){
        bool draw_pie = true;
        int pie[6] = { 0 };
        const char* pie_sm = "c_sm.csv";
        FILE *pie_c_sm = fopen(pie_sm,"aw");
        if (draw_pie){
            for ( auto &p:c_sm ){
                if ( p.second==1 ){
                    pie[0]++;
                }else if ( p.second==2 ){
                    pie[1]++;
                }else if ( p.second==3 ){
                    pie[2]++;
                }else if ( p.second>3 && p.second<=5 ){
                    pie[3]++;
                }else if ( p.second>5 && p.second<=10 ){
                    pie[4]++;
                }else{
                    pie[5]++;
                }
            }
        }
        fprintf(pie_c_sm, "%ld,",uni_nb);
        for (int ii=0; ii<6; ++ii){
            if (ii<5) fprintf(pie_c_sm, "%d,",pie[ii]);
            else fprintf(pie_c_sm, "%d\n",pie[ii]);
        }
        fclose(pie_c_sm);
    }

    bool collect_b_loads = false;
    bool collect_ops = true;
    bool collect_tile_alloc = false;
    const char* l1cache = "l1cache.csv";
    if (collect_b_loads){
        l1cache = "b_loads_per_sm.csv";
    }
    if (collect_ops){
        l1cache = "ops_per_sm.csv";
    }
    if (collect_tile_alloc){
        l1cache = "tiles_per_sm.csv";
    }
    FILE *l1_est = fopen(l1cache,"aw");
    fprintf(l1_est,"%s,",dl.graph_name.c_str());
    fprintf(l1_est,"%s,",dl.vertex_order_abbr.c_str());
    fprintf(l1_est,"%d\n",tm);
    if (validate_nnz!=nnz){
        printf("%d of %s, val = %d, nnz = %d\n",__LINE__,__FILE__,(int)validate_nnz,(int)nnz);
    }
    assert(validate_nnz==nnz);
    int validate_segs = n_segs;
    for (int j=0; j<sms; ++j){
        acc_col += col_st[j].size();
        if (collect_b_loads){
            fprintf(l1_est,"%d,", (int)col_st[j].size());
        }
        if (collect_ops){
            int nnz_in_sm = 0;
            if ( next_seg[ j ]<grouped_tailSeg[ j ] ){
                nnz_in_sm = seg_rowPtr[grouped_tailSeg[j]*(tm+1)-1] - seg_rowPtr[next_seg[j]*(tm+1)];
                int tiles_in_sm = grouped_tailSeg[j] - next_seg[j];
                assert(tiles_in_sm>0);
            }
            fprintf(l1_est,"%d,", nnz_in_sm);
        }
        if (collect_tile_alloc){
            fprintf(l1_est,"%d,", grouped_tailSeg[j]-next_seg[j]);
            validate_segs -= (grouped_tailSeg[j]-next_seg[j]);
        }
    }
    if (collect_b_loads){
        fprintf(l1_est,"%d\n", (int)col_st[ sms ].size());
    }
    if(collect_ops){
        if ( next_seg[ sms ]<grouped_tailSeg[ sms ] ){
            fprintf(l1_est,"%d\n", seg_rowPtr[grouped_tailSeg[sms]*(tm+1)-1]-
                                                seg_rowPtr[next_seg[sms]*(tm+1)] );
        }else{
            fprintf(l1_est,"%d\n", 0);
        }
    }
    if (collect_tile_alloc){
        fprintf(l1_est,"%d\n", grouped_tailSeg[ sms ]-next_seg[ sms ]);
        validate_segs -= (grouped_tailSeg[ sms ]-next_seg[ sms ]);
        assert( validate_segs==0 );
    }
/******************************************************************************/

    est_fp = int64_t(nnz)*k;
    // shadow_b_bytes is identical to gpuX_bytes when perform v9
    // so dl.gpuX_bytes can be seen shadow_b_bytes when v9

    //int64_t est_ld_bytes1 = int64_t(segNzRowIdx_bytes) + segNzColIdx_bytes + 
    //                vals_bytes + segPtr_bytes; 
    
    int64_t est_ld_bytes2 = int64_t(segNzCV_bytes) + seg_rowPtr_bytes;
    
    raw_ld_bytes = vals_bytes +
                   dl.gpuX_bytes;

    est_ld_bytes = est_ld_bytes2 +
                   segVoMap_bytes +
                   dl.gpuX_bytes +
                  grouped_tailSeg_bytes +
                   next_seg_bytes;    
        
    est_ld_bytes_tiling_ideal = est_ld_bytes2 +
                                segVoMap_bytes +
                                n_col_sum*k*4 +
                                grouped_tailSeg_bytes +
                                next_seg_bytes;    
    
    est_ld_bytes_tiling_sm_ideal = est_ld_bytes2 +
                                   segVoMap_bytes +
                                   acc_col*k*4 +
                                   grouped_tailSeg_bytes +
                                   next_seg_bytes;    

    // acc_col should be less than n_col_sum
    if (false)  printf("%d of %s, n_col_sum = %ld, acc_col = %ld\n",__LINE__,__FILE__,n_col_sum,acc_col);
    
    est_st_bytes = dl.gpuC_bytes;
}
void Mat::alpha_transfer(){
#   define CMALC(var)                                   \
     var##_bytes = var.size() * sizeof( var[0] );        \
     CHECK_CUDA(hipMalloc( &var##_dev, var##_bytes )) ;

     CMALC( alpha_rowPtr ); CMALC( alpha_colIdx ); CMALC( alpha_vals ); 
     CMALC( alpha_pillar_rowPtr ); CMALC( alpha_pillarIdx ); CMALC( segVoMap ); 
     CMALC( voMp ); 
#   undef CMALC

    // transfer data to device
    hipMemcpy(alpha_rowPtr_dev, alpha_rowPtr.data(), alpha_rowPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(alpha_colIdx_dev, alpha_colIdx.data(), alpha_colIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(alpha_pillar_rowPtr_dev, alpha_pillar_rowPtr.data(), alpha_pillar_rowPtr.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(alpha_vals_dev, alpha_vals.data(), alpha_vals.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(alpha_pillarIdx_dev, alpha_pillarIdx.data(), alpha_pillarIdx.size()*sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(segVoMap_dev, segVoMap.data(), segVoMap.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(voMp_dev, voMp.data(), voMp.size()*sizeof(int), hipMemcpyHostToDevice);
    if (dl.vertex_order_abbr != "OVO"){
        CHECK_CUDA(hipMalloc( &shadow_b_dev,  m*k*sizeof(float))) ;
        CHECK_CUDA(hipMemset( shadow_b_dev,  0, m*k*sizeof(float))) ;
    }
    CHECK_CUDA(hipMalloc( &counter_dev,  (sms+1)*sizeof(int))) ;
    CHECK_CUDA(hipMemset( counter_dev,  0, (sms+1)*sizeof(int))) ;
}
void Mat::alpha_dataVolume_est(){
}
void Mat::dataVolume_est(){
    est_fp = int64_t(nnz)*k;
    // shadow_b_bytes is identical to gpuX_bytes when perform v9
    // so dl.gpuX_bytes can be seen shadow_b_bytes when v9
    est_ld_bytes = int64_t(tileNnz_bytes) + 
                    tileColIdx_bytes + 
                    vals_bytes + 
                    dl.gpuX_bytes +
                    tileRowPtr_bytes + 
                    nnzTile_bytes + 
                    bitMap_bytes + 
                    rcOffset_bytes +
                    voMp_bytes;
    est_st_bytes = dl.gpuC_bytes;
}
void Mat::permute_segs(){
	std::vector<unsigned int> segPtr1(1,0);
	std::vector<unsigned int> segNzRCIdx1;
	std::vector<float> newVals1;
	std::vector<unsigned int> segVoMap1;

    // {seg row_idx,seg_idx}
	std::pair last{-1,-1};

	while (!aux_seg.empty()){
		auto top = aux_seg.front();
		aux_seg.pop();
		
		if ( count_segs[top.first] != aux_seg.size()+1 && last.first!=-1 && top.first == last.first ){
			aux_seg.push(top);
		}else{
            count_segs[top.first]--;
			int seg_idx = top.second;
            int seg_nnz = segPtr[seg_idx+1] - segPtr[seg_idx];
			for (int i=segPtr[seg_idx]; i<segPtr[seg_idx+1]; ++i){
				segNzRCIdx1.push_back(segNzRCIdx[2*i]);
				segNzRCIdx1.push_back(segNzRCIdx[2*i+1]);
				newVals1.push_back(newVals[i]);
			}
			segPtr1.push_back(segPtr1.back()+seg_nnz);
			for (int i=0; i<tm; ++i){
				segVoMap1.push_back(segVoMap[seg_idx*tm+i]);
			}

			last = top;
		}
	}
	swap(segPtr, segPtr1);
	swap(segNzRCIdx, segNzRCIdx1);
	swap(newVals, newVals1);
	swap(segVoMap, segVoMap1);
	return ;
}
int Mat::checkSim(vector<int>& a, vector<int>& b){
    // check the number of colnum overlap (non-zeros)
    // to improve (temporal) locality of dense input in L1 
    int sim = 0;
    int i = 0;
    int j = 0;
    while ( i<a.size() && j<b.size() ){
        if ( a[i]<b[j] ){
            i++;
        }else if ( a[i]>b[j] ){
            j++;
        }else{
            sim++;i++;j++;
        }
    }
    return sim;
}
void Mat::dfsSegs(){

    unordered_set<int> insular;
    // construct graph
    // { idx, col overlaps } min heap, sort by col overlaps
    // enable the max col overlap be on the top of the stack when DFS
    vector< priority_queue<pair<int,int>, vector<pair<int,int>>, cmp> >
      g(n_segs);

    vector< vector<int> > col_to_seg(n);
    for (int i=0; i<n_segs; i++ )
      for ( auto c: cols_seg[i] ) col_to_seg[c].push_back(i);

    vector<int> mark(n_segs,-1);

    for (int i=0; i<n_segs; ++i)
      {
        for ( auto col_i: cols_seg[i] )
          for ( auto seg_j: col_to_seg[col_i] )
            {
              // 
              if ( seg_j <= i ) continue;
              // pruning, two segs cannot be on the same row panel
              if ( id2r[i] == id2r[seg_j] ) continue; 
              // pruning, check if the seg_j-th seg has been paired
              if ( mark[seg_j] == i ) continue;
              mark[seg_j] = i;
              int sim = checkSim(cols_seg[i],cols_seg[seg_j]);
              if ( sim ){
                g[i].push({seg_j, sim});
                g[seg_j].push({i, sim});
              }
            }
        if ( g[i].empty() ) {
          insular.insert(i);
          g[i].push( {i,0} );
        }
      }

    if ( insular.size()>0 ){
        printf("insular segs = %lu\n",insular.size());
        //assert( insular.size()==0 );
    }
	std::vector<unsigned int> segPtr1(1,0);
	std::vector<unsigned int> segNzRCIdx1;
	std::vector<unsigned int> segNzRowIdx1;
	std::vector<unsigned int> segNzRowIdx_2bit1;
	std::vector<unsigned int> segNzColIdx1;
	std::vector<float> newVals1;
	std::vector<unsigned int> segVoMap1;

	std::vector<float> segNzCV1;
	std::vector<int> seg_rowPtr1;
    // DFS reorder segs
    // explore L1 reuse
    vector<bool> visited(n_segs,false);
    stack<int> st;
    unsigned val_seg = 0;
    unsigned val_w = 0;
    for (int src=0; src<n_segs; ++src){
        if (visited[src] || insular.find(src)!=insular.end())   continue;
        st.push(src);
        while ( !st.empty() ){
            
            int node = st.top();
            st.pop();
            if (visited[node])  continue;
            visited[node] = true;
            val_seg++;

            int seg_nnz = segPtr[node+1] - segPtr[node];
            val_w += seg_nnz;
            for (int i=segPtr[node]; i<segPtr[node+1]; ++i){
                segNzRCIdx1.push_back(segNzRCIdx[2*i]);
                segNzRowIdx1.push_back(segNzRCIdx[2*i]);
                segNzRCIdx1.push_back(segNzRCIdx[2*i+1]);
                segNzColIdx1.push_back(segNzRCIdx[2*i+1]);
                
                newVals1.push_back(newVals[i]);
                
                segNzCV1.push_back(segNzCV[2*i]);
                segNzCV1.push_back(segNzCV[2*i+1]);
        
            }        
            segPtr1.push_back(segPtr1.back()+seg_nnz);
            for (int i=0; i<tm; ++i){
                segVoMap1.push_back(segVoMap[node*tm+i]);
            }
            
            if ( seg_rowPtr1.empty() )  seg_rowPtr1.push_back(0);
            else    seg_rowPtr1.push_back( seg_rowPtr1.back() );
            for (int i=0; i<tm; ++i){
                seg_rowPtr1.push_back( seg_rowPtr1.back() + (seg_rowPtr[node*(tm+1)+i+1] - seg_rowPtr[node*(tm+1)+i]) );
            }
            
            while ( !g[node].empty() ){

                auto nb = g[node].top();
                g[node].pop();
                if ( !visited[nb.first] ){
                    st.push(nb.first);
                }
            }            
        }
    }
    for (int node:insular){
    
        int seg_nnz = segPtr[node+1] - segPtr[node];
        for (int i=segPtr[node]; i<segPtr[node+1]; ++i){
            segNzRCIdx1.push_back(segNzRCIdx[2*i]);
            segNzRowIdx1.push_back(segNzRCIdx[2*i]);
            segNzRCIdx1.push_back(segNzRCIdx[2*i+1]);
            segNzColIdx1.push_back(segNzRCIdx[2*i+1]);
            
            newVals1.push_back(newVals[i]); 
            
            segNzCV1.push_back(segNzCV[2*i]);
            segNzCV1.push_back(segNzCV[2*i+1]);
        } 
        segPtr1.push_back(segPtr1.back()+seg_nnz);
        for (int i=0; i<tm; ++i){
            segVoMap1.push_back(segVoMap[node*tm+i]);
        }
       
        if ( seg_rowPtr1.empty() )  seg_rowPtr1.push_back(0);
        else    seg_rowPtr1.push_back( seg_rowPtr1.back() );
        for (int i=0; i<tm; ++i){
            seg_rowPtr1.push_back( seg_rowPtr1.back() + (seg_rowPtr[node*(tm+1)+i+1] - seg_rowPtr[node*(tm+1)+i]) );
        }
    }

	assert( segPtr.size()==segPtr1.size() );
	assert( segNzRCIdx.size()==segNzRCIdx1.size() );
	assert( segNzRowIdx.size()==segNzRowIdx1.size() );
	assert( segNzColIdx.size()==segNzColIdx1.size() );
	assert( newVals.size()==newVals1.size() );
	assert( segVoMap.size()==segVoMap1.size() );
    assert( segNzCV.size()==segNzCV1.size() );
    assert( seg_rowPtr.size()==seg_rowPtr1.size() );
      
	swap(segPtr, segPtr1);
	swap(segNzRCIdx, segNzRCIdx1);
	swap(segNzRowIdx, segNzRowIdx1);
	swap(segNzColIdx, segNzColIdx1);
	swap(newVals, newVals1);
	swap(segVoMap, segVoMap1);
    swap(segNzCV, segNzCV1);
    swap(seg_rowPtr, seg_rowPtr1);
}
int Mat::checkSim2(map<int,int>& a, vector<int>& b){
    // check the number of colnum overlap (non-zeros)
    // to improve (temporal) locality of dense input in L1 
    int sim = 0;
    int j = 0;
    while ( j<b.size() ){
        if ( a.find(b[j++])!=a.end() ){
            sim++;
        }
    }
    return sim;
}
void Mat::sliWinSegs(){

    vector<int> insular;

    vector< vector<int> > col_to_seg(n);
    for (int i=0; i<n_segs; i++ )
      for ( auto c: cols_seg[i] ) col_to_seg[c].push_back(i);


	std::vector<unsigned int> segPtr1(1,0);
	std::vector<unsigned int> segNzRCIdx1;
	std::vector<unsigned int> segNzRowIdx1;
	std::vector<unsigned int> segNzRowIdx_2bit1;
	std::vector<unsigned int> segNzColIdx1;
	std::vector<float> newVals1;
	std::vector<unsigned int> segVoMap1;

	std::vector<float> segNzCV1;
	std::vector<int> seg_rowPtr1;
    // DFS reorder segs
    // explore L1 reuse
    vector<bool> visited(n_segs,false);
    
    vector<int> sh_seg_id(n_segs);
    std::iota(sh_seg_id.begin(),sh_seg_id.end(),0);
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine eee(seed);
    std::shuffle(sh_seg_id.begin(), sh_seg_id.end(), eee);

    vector<int> seg_ord;
    int window = 64; // expected active_warps 
    for (int src:sh_seg_id){
        if ( visited[src] )   continue;
        visited[src] = true;
       
        // {col_idx, freq}, like a sliding window to kepp track of the latest #window segs 
        map<int,int> col_in_cache;
        std::transform( std::begin(cols_seg[src]),std::end(cols_seg[src]),
                std::inserter(col_in_cache, col_in_cache.end()),
                [](int colID) {return std::make_pair(colID,1);} ); 
        vector<int> tree_seg_ord;
        // while loop for current graph component traversal
        int val_seg = 0;
        while (true)
        { 
            val_seg++;
            assert(val_seg<=n_segs); // just to detect infinite loop

            int candidate_seg = src;

            // at least 1 col overlap between two tiles
            int mx_sim = 1;
            // explore the tile having maximum col overlaps 
            // with previous #window tiles
            for ( auto col_i: col_in_cache )
              for ( auto seg_j: col_to_seg[col_i.first] ) // all segs having col_i.first
                {
                  if ( visited[seg_j] )   continue;
                  int sim = checkSim2(col_in_cache,cols_seg[seg_j]);
                  if ( sim>mx_sim ){
                      mx_sim = sim;
                      candidate_seg = seg_j;
                  }
                }
            // no matter it is insular or new seg
            visited[candidate_seg] = true;
            // check if we find a potential to grow the tree 
            if(candidate_seg!=src){
                // found one potential
                // update col_in_cache
                // works like a sliding window
                if (tree_seg_ord.size()>=window){
                    int to_be_evict = *(tree_seg_ord.rbegin()+window-1);
                    for ( auto col_ii: cols_seg[to_be_evict] ){
                        if (--col_in_cache[col_ii]==0){
                           col_in_cache.erase(col_ii); 
                        }
                    }
                }
                for ( auto col_ii: cols_seg[candidate_seg] ){
                    col_in_cache[col_ii]++; 
                }

                // push the candidate into seg_ord
                if (tree_seg_ord.size()==0) tree_seg_ord.push_back(src);
                tree_seg_ord.push_back(candidate_seg); 
            }else{
                // leaf OR insular
                if (tree_seg_ord.size()==0){
                    // insular
                    insular.push_back(src);
                }
                break;
            }
        }
        // merge the current tree into our forest
        seg_ord.insert(seg_ord.end(),tree_seg_ord.begin(),tree_seg_ord.end()); 

    }
    if (false)
    {
        printf("%d of %s : #segs = %d\n",__LINE__,__FILE__,n_segs);
        printf("%d of %s : #seg_ord = %lu, #insular = %lu\n",__LINE__,__FILE__,
                seg_ord.size(),insular.size());
    }
    // merge the current tree into our forest
    seg_ord.insert(seg_ord.end(),insular.begin(),insular.end()); 
    assert(seg_ord.size()==n_segs);
    for (int node:seg_ord){
    
        int seg_nnz = segPtr[node+1] - segPtr[node];
        for (int i=segPtr[node]; i<segPtr[node+1]; ++i){
            segNzRCIdx1.push_back(segNzRCIdx[2*i]);
            segNzRowIdx1.push_back(segNzRCIdx[2*i]);
            segNzRCIdx1.push_back(segNzRCIdx[2*i+1]);
            segNzColIdx1.push_back(segNzRCIdx[2*i+1]);
            
            newVals1.push_back(newVals[i]); 
            
            segNzCV1.push_back(segNzCV[2*i]);
            segNzCV1.push_back(segNzCV[2*i+1]);
        } 
        segPtr1.push_back(segPtr1.back()+seg_nnz);
        for (int i=0; i<tm; ++i){
            segVoMap1.push_back(segVoMap[node*tm+i]);
        }
       
        if ( seg_rowPtr1.empty() )  seg_rowPtr1.push_back(0);
        else    seg_rowPtr1.push_back( seg_rowPtr1.back() );
        for (int i=0; i<tm; ++i){
            seg_rowPtr1.push_back( seg_rowPtr1.back() + (seg_rowPtr[node*(tm+1)+i+1] - seg_rowPtr[node*(tm+1)+i]) );
        }
    }

	assert( segPtr.size()==segPtr1.size() );
	assert( segNzRCIdx.size()==segNzRCIdx1.size() );
	assert( segNzRowIdx.size()==segNzRowIdx1.size() );
	assert( segNzColIdx.size()==segNzColIdx1.size() );
	assert( newVals.size()==newVals1.size() );
	assert( segVoMap.size()==segVoMap1.size() );
    assert( segNzCV.size()==segNzCV1.size() );
    assert( seg_rowPtr.size()==seg_rowPtr1.size() );
      
	swap(segPtr, segPtr1);
	swap(segNzRCIdx, segNzRCIdx1);
	swap(segNzRowIdx, segNzRowIdx1);
	swap(segNzColIdx, segNzColIdx1);
	swap(newVals, newVals1);
	swap(segVoMap, segVoMap1);
    swap(segNzCV, segNzCV1);
    swap(seg_rowPtr, seg_rowPtr1);
}

void Mat::csr2_DiagTiling(){
    int device_id;
    hipDeviceProp_t prop;
    hipGetDevice( &device_id );
    hipGetDeviceProperties( &prop, device_id );
    int n_sm = prop.multiProcessorCount;
    sms = n_sm;
    // int n_sm = 114;
    const int warps_per_sm = 64; // 32 blocks per SM
    const int wing_tiles = 10;
    float alpha = 0.3;
    /*create tiles along the diagonal band (lower & upper diagnonal)*/ 

    // I assume 20 percent of nz/weights will be covered in diagonal tiles
    const int nnz_diagonal_tiles = alpha * rowPtr[m]; 
    
    const int partitions_node = warps_per_sm * n_sm; // partitions along the width
    const int nnz_p_diagonal_tile  = max(32,nnz_diagonal_tiles / partitions_node); 
    //printf("nnz_diagonal_tiles = %d, nnz_p_diagonal_tile = %d\n",nnz_diagonal_tiles,nnz_p_diagonal_tile);
    vector<int> tile_width(partitions_node,0);
    // In the first round, we conly construct the diagonal tiles, whcih act as the central tiles
    // for each SM. The diagonal tiles are constructed by exploring the row panel and col panel
    // Each SM will have at least one tile
    int mat_r_start = 0;
    int warps_with_weights = 0;
    for (int i=0; i<partitions_node; ++i){
        mat_r_start += i? tile_width[i-1] : 0;
        // not necessary to let all SMs have non-zeros/weights. e.g. assert(mat_r_start<m-1); 
        // These SMs can process the last bucket directly
        int nnz_current_diag_tile = 0;
        int j = mat_r_start;
        while (j<m && nnz_current_diag_tile<=(int)(0.9*nnz_p_diagonal_tile)){
            // explore row_panel
            for (int k=rowPtr[j]; colIdx[k]<=j; ++k){
                if (colIdx[k]>=mat_r_start){
                    nnz_current_diag_tile++;
                }
                if (colIdx[k]==j){
                    break;
                }
            }
            // explore col_panel
            for (int k=mat_r_start; k<j; ++k){  // k is the row idx
                int l=rowPtr[k]; 
                while (l < rowPtr[k+1] && colIdx[l]<j){
                    l++;
                }
                nnz_current_diag_tile += (colIdx[l]==j);
            }

            j++;
            
        }
        warps_with_weights += (nnz_current_diag_tile>0);
        tile_width[i] = j - mat_r_start;
        //printf("tile_width[%d] = %d, nnz_current_diag_tile = %d\n",i,tile_width[i],nnz_current_diag_tile);
    }
    int verify_m = accumulate(tile_width.begin(), tile_width.end(), 0);
    if (verify_m<m && tile_width.back()>0){
        printf("alpha is too small ...  verify_m = %d, m = %d\n",verify_m,m);
    }
    assert(verify_m==m);
    // In the second round, we extend twenty more tiles for each diagonal tile.
    // ten above the diagonal and ten below the diagonal
    // weights/non-zeros for each warp are stored in CSC
    vector<int> nnz_p_warp(partitions_node+1,0);
    int nnz_rowPtr = 0;
    int col_accu_start = 0;
    alpha_pillar_rowPtr.push_back(0);
    
    for (int i=0; i<warps_with_weights; ++i){
        
        int col_start = col_accu_start;
        int col_end = col_accu_start;
        if (i>=wing_tiles) col_accu_start += tile_width[i-wing_tiles]; 

        for (int j=wing_tiles; j>0; --j){
            if (i-j>=0){
                col_end += tile_width[i-j];
            }    
        }
        int row_start = col_end; // the diagonal tile is square
        
        col_end += tile_width[i];
        for (int j=1; j<=wing_tiles; ++j){
            if (i+j<warps_with_weights){
                col_end += tile_width[i+j];
            }else{
                break;
            }
        }
        
        int row_end = row_start + tile_width[i];
        
        for (int j=row_start; j<row_end; ++j){
            // visit each col_panel
            int entries_in_row = 0;
            alpha_rowPtr.push_back(nnz_rowPtr);
            
            for (int kk=rowPtr[j]; kk<rowPtr[j+1]; ++kk){  
                int l=colIdx[kk];
                if (l<col_start){
                    continue;
                }
                if (l>=col_end){
                    break;
                }
                
                //printf("r = %d, c = %d, val[%d] = %f\n",j,l,k,vals[k]);
                alpha_colIdx.push_back(l);
                alpha_vals.push_back(vals[kk]);
                entries_in_row++;
                nnz_p_warp[i]++;
                nnz_rowPtr++;
            }
            if ( entries_in_row>=0 && entries_in_row<(rowPtr[j+1]-rowPtr[j]) ){
                // if the #nz in a specific row of a seg 
                // is less than that of the whole row,
                // the row requires "atomic add".
                // use MSB to mark it.
                segVoMap.push_back( voMp[j] | (1<<31) );
            }else{ 
                segVoMap.push_back( voMp[j] );
            }
        }
        if (nnz_p_warp[i]){
            alpha_pillar_rowPtr.push_back(alpha_pillar_rowPtr.back() + tile_width[i]);
        } 
        if (i%warps_per_sm==0){
            alpha_pillarIdx.push_back(i);
        }
        //printf("nnz_p_warp[%d] = %d\n",i,nnz_p_warp[i]);
    }
    while ( alpha_pillarIdx.size()<=n_sm ) alpha_pillarIdx.push_back(warps_with_weights);

    assert(nnz_rowPtr<=rowPtr[m]);
    // if there is an SM has no non-zero/weight,the following assert will fail
    if (alpha_rowPtr.size()!=m){
        printf("alpha_rowPtr.size() = %zu\n",alpha_rowPtr.size());
        printf("(m+warps_with_weights) = %d\n",(m+warps_with_weights));
    }
    assert(alpha_rowPtr.size()==m);

    empty_wp_p = (1- (float)warps_with_weights/partitions_node)*100;
    band_nz_p = (float)alpha_colIdx.size()/rowPtr[m]*100;      // non-zero percentage of non-balance workloads. can be tuned by adjusting wing_tiles.

    // In the third round, we fill in the last bucket with the remaining weights/non-zeros.
    // It can be merged with the second round, but I just want to keep it simple
    // and easy to understand.
    // The last bucket is used for workload balance among SMs
    col_accu_start = 0;
    int pillars_in_total = warps_with_weights;
    for (int i=0; i<warps_with_weights; ++i){
        int col_start = col_accu_start;
        int col_end = col_accu_start;
        if (i>=wing_tiles) col_accu_start += tile_width[i-wing_tiles]; 
        
        for (int j=wing_tiles; j>0; --j){
            if (i-j>=0){
                col_end += tile_width[i-j];
            }    
        }
        int row_start = col_end; // the diagonal tile is square
        col_end += tile_width[i];
        for (int j=1; j<=wing_tiles; ++j){
            if (i+j<warps_with_weights){
                col_end += tile_width[i+j];
            }else{
                break;
            }
        }  
        
        int row_end = row_start + tile_width[i];
        int nnz_current_pillar = 0;
        vector<int> temp_alpha_rowPtr;
        vector<int> temp_segVoMap;
        for (int j=row_start; j<row_end; ++j){
            
            // visit each col_panel
            int entries_in_row = 0;
            temp_alpha_rowPtr.push_back(nnz_rowPtr);
            
            for (int kk=rowPtr[j]; kk<rowPtr[j+1]; ++kk){  // k is the row idx
                int l=colIdx[kk];
                if (l>=col_start && l<col_end){
                    continue;
                }
                
                //printf("r = %d, c = %d, val[%d] = %f\n",j,l,k,vals[k]);
                assert(alpha_colIdx.size()==alpha_vals.size() && alpha_vals.size()==nnz_rowPtr);
                alpha_colIdx.push_back(l);
                alpha_vals.push_back(vals[kk]);
                entries_in_row++;
                nnz_p_warp[i]++;
                
                nnz_rowPtr++;
                nnz_current_pillar++;
            }
            
            if ( entries_in_row>=0 && entries_in_row<(rowPtr[j+1]-rowPtr[j]) ){
                // if the #nz in a specific row of a seg 
                // is less than that of the whole row,
                // the row requires "atomic add".
                // use MSB to mark it.
                temp_segVoMap.push_back( voMp[j] | (1<<31) );
            }else{ 
                temp_segVoMap.push_back( voMp[j] );
            }  
        }
        
        if (nnz_current_pillar){
            for (auto & rid: temp_alpha_rowPtr){
                alpha_rowPtr.push_back(rid);
            }
            for (auto & vm: temp_segVoMap){
                segVoMap.push_back(vm);
            }
            pillars_in_total++;
            alpha_pillar_rowPtr.push_back(alpha_pillar_rowPtr.back() + tile_width[i]);
        }
    }
   
    alpha_rowPtr.push_back(nnz_rowPtr);
    alpha_pillarIdx.push_back(pillars_in_total);
    
    n_segs = alpha_pillarIdx.back();

    if ( alpha_colIdx.size()!=alpha_vals.size() ||
         alpha_vals.size()!=nnz_rowPtr ||
         nnz_rowPtr!=rowPtr[m] ||
         alpha_pillarIdx.size()!=n_sm+2 ||
         segVoMap.size()+1!=alpha_rowPtr.size()){
        printf("alpha_colIdx.size() = %zu, alpha_vals.size() = %zu, nnz_rowPtr = %d\n",
                alpha_colIdx.size(),alpha_vals.size(),nnz_rowPtr);
        printf("alpha_pillarIdx.size() = %zu, expect = %d\n",alpha_pillarIdx.size(),n_sm+2);
        printf("segVoMap.size()+1 = %zu, alpha_rowPtr.size() = %zu\n",segVoMap.size()+1,alpha_rowPtr.size());
        
    }
    assert(alpha_colIdx.size()==alpha_vals.size());
    assert(alpha_vals.size()==nnz_rowPtr);
    assert(nnz_rowPtr==rowPtr[m]);
    // first #SM entries mark the start pillar index for each SM queue.
    // The penultimate marks the start pillar for workload balance. 
    // The last marks the next of the end.(#row pillars in total)
    assert(alpha_pillarIdx.size()==n_sm+2); 
    assert(segVoMap.size()+1==alpha_rowPtr.size()); 
}

void
Mat::alpha_stats_collect(FILE *stream)
{
  
  const uint n_pillars = alpha_pillar_rowPtr.size()-1;

  
  // expect perfect reuse within a pillar
  n_col_sum = 0;
  for ( uint pillar_idx = 0; pillar_idx < n_pillars; pillar_idx++ )
    {
      unordered_set<int> colset; 
      for (int i=alpha_rowPtr[alpha_pillar_rowPtr[pillar_idx]]; i<alpha_rowPtr[alpha_pillar_rowPtr[pillar_idx+1]]; ++i){
        colset.insert(alpha_colIdx[i]);
      }
      n_col_sum += colset.size();
    }

  // expect perfect reuse within an SM
  // Assume workloads in the last queue (used for workload balance) are processed in an SM
  acc_col = 0;
  for (uint pillars_sm = 0; pillars_sm < alpha_pillarIdx.size()-1; ++pillars_sm){
    int start = alpha_pillarIdx[pillars_sm];
    int end = alpha_pillarIdx[pillars_sm+1];
    unordered_set<int> colset;
    for (uint j=alpha_rowPtr[alpha_pillar_rowPtr[start]]; j<alpha_rowPtr[alpha_pillar_rowPtr[end]]; ++j){
      colset.insert(alpha_colIdx[j]);
    }
    acc_col += colset.size();
  }

  for ( uint32_t v: segVoMap ) if ( v & (1u<<31) ) atomic_op++;

}

void Mat::csr2tile(){


  const int nnz_csr = rowPtr[m];

    bool print_bucket = false;
	int tileRows = (m+tm-1)/tm;
    // const char* tiles_per_row_panel = "tiles_per_row_panel.csv";
    // FILE *tile_nperf = fopen(tiles_per_row_panel,"aw");
    vector<int> segs_per_row_panel(tileRows,0);
	for (int i=0; i<tileRows; ++i){
		//csr2flex_Rmajor(i);
		//csr2flex_Cmajor(i);
		//csr2regular(i);
        int temp = segPtr.size();
        csr2seg_Cmajor(i);
        segs_per_row_panel[i] = segPtr.size()-temp;
	} 

    assert( nnz_csr == seg_rowPtr.back() );

    n_segs = segPtr.size()-1;
    if (print_bucket) printf("%d of %s, n_segs = %d\n",__LINE__, __FILE__, n_segs); 
    bool seg_sort = false;
    if (seg_sort) {
        //permute_segs();
        //dfsSegs();
       sliWinSegs();
    }
    
        int device_id;
        hipDeviceProp_t prop;
        hipGetDevice( &device_id );
        hipGetDeviceProperties( &prop, device_id );
        int n_sm = prop.multiProcessorCount;
        sms = n_sm; 
        
        // distribute segs into n_sm+1 buckets, contiguous segs are in a bucket
        // according to #non zeros ( wkload per sm )
        // to balance workload, the last bucket is to offer segs when faster SMs are free   
        int nnz = newVals.size(); 

        assert( nnz == nnz_csr );

        int wkload = nnz / n_sm; 
        int segload = n_segs / n_sm;
        int seg_head_sm = 0;
        int seg_tail_sm = 0;
        int validate_nnz = 0;
        
        // assign segs to each sm bucket
        bool nnz_based_split = false;
        bool row_based_split = true;
        int panel_idx = 0;
        for (int i=0; i<n_sm; ++i){
            next_seg.push_back( seg_head_sm );
            int nz = 0;
            
            if (nnz_based_split){
                nz = segPtr[seg_head_sm+1] - segPtr[seg_head_sm];
                seg_tail_sm = seg_head_sm + 1;
                while ( seg_tail_sm < n_segs && nz<(int)(0.98*wkload) ){
                    nz += (segPtr[seg_tail_sm+1] - segPtr[seg_tail_sm]);
                    seg_tail_sm++;
                }
            }
            if (row_based_split && (panel_idx<tileRows) ){
                int current_bin_num_segs = segs_per_row_panel[panel_idx];  // current row panel
                seg_tail_sm = seg_head_sm + current_bin_num_segs;
                while ( ++panel_idx < tileRows ){
                    if ( (segs_per_row_panel[panel_idx] + current_bin_num_segs) > segload ){
                        break;
                    }
                    current_bin_num_segs += segs_per_row_panel[panel_idx];
                    seg_tail_sm += segs_per_row_panel[panel_idx];
                }
                nz += (segPtr[seg_tail_sm] - segPtr[seg_head_sm]);
            }
            // printf("#sm = %d, panelID = %d / %d,seg_head_sm = %d, seg_tail_sm = %d, nz = %d / %d\n",i,panel_idx, tileRows, seg_head_sm,seg_tail_sm,nz,segPtr.back());
            validate_nnz += nz;
            grouped_tailSeg.push_back( min(n_segs,seg_tail_sm) );
            if ( seg_head_sm==min(n_segs,seg_tail_sm) ){
                empty_bucket++;
            }
            seg_head_sm = min(n_segs,seg_tail_sm);
        }
        
        // the last bucket is used for workload balance among SMs 
        // if seg_head_sm==n_segs, then n_segs==seg_head_sm
        next_seg.push_back( seg_head_sm );
        grouped_tailSeg.push_back( n_segs );
        validate_nnz += segPtr[n_segs]-segPtr[seg_head_sm];
        assert( validate_nnz==segPtr.back() );
        assert( grouped_tailSeg.size()==n_sm+1 );
        assert( next_seg.size()==n_sm+1 );
        //printf("empty_bucket = %d, balance_segs = %d, balance_row_panels = %d\n",empty_bucket, n_segs-seg_head_sm, tileRows-panel_idx);
}
void Mat::print3(int l){
    if ( true ){
        printf("\nSegPtr: \n");
        for (int i=0; i<(l?l:segPtr.size()); ++i){
            printf("(%d:%d)  ",i,segPtr[i]);
        }
        printf("\nSegNzRC: \n");
        for (int i=0; i<(l?l:segNzRCIdx.size()/2); ++i){
            printf("(%d:%d)  ",segNzRCIdx[2*i],segNzRCIdx[2*i+1]);
        }
        if (false){
            printf("\nSegRowNzIdx: %d\n",(int)segNzRowIdx.size());
            for (int i=0; i<(l?l:segNzRowIdx.size()); ++i){
                printf("%d  ",segNzRowIdx[i]);
            }
            printf("\nSegColNzIdx: %d\n",(int)segNzColIdx.size());
            for (int i=0; i<(l?l:segNzColIdx.size()); ++i){
                printf("%d  ",segNzColIdx[i]);
            }
        }
    }
    printf("\nSegVoMap: %d\n",(int)segVoMap.size());
    for (int i=0; i<(l?l:segVoMap.size()); ++i){
        printf("%d->%d  ",i,segVoMap[i]&0x7fffffff);
    }
    printf("\n");
}

void Mat::csr2seg_Cmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

	// keep track of the cols in each row
	std::vector<int> cOffset(tm, 0);

    // {col, val}, for kernel v31 
    std::vector<std::vector<std::pair<int,float>>> segcv(tm, std::vector<std::pair<int,float>>()); 


    int dif = 0.1*nnz_limit; 
    int nnzInSeg = 0;
    int nnz_cur_panel = rowPtr[rowEnd] - rowPtr[rowStart];    

    // If n_nodes_z_out>0 some panels can be empty, which tiling can't handle.
    assert( !dl.dl_original->n_nodes_z_out );
    vector<int> atom(tm, 0);

    map<int,int> occ_cols;
    for ( auto c: views::iota(rowPtr[rowStart],rowPtr[rowEnd]) )
      occ_cols[colIdx[c]]++;
    const auto last_col = occ_cols.rbegin()->first;
    // collect segs in the panel
    for ( auto [j,ncol]: occ_cols ) {
        
        int segId = segPtr.size()-1;
        for ( int i=rowStart; i<rowEnd; ++i ){
            // absolute position of the nze in csr, idx = base + offset
            int c = rowPtr[i] + cOffset[i-rowStart];
            if ( colIdx[c]==j && c<rowPtr[i+1] ){
                // nze values
                segNzRowIdx.push_back(i-rowStart);
                segNzColIdx.push_back(j);
                
                segcv[i-rowStart].push_back({j,vals[c]}); // for v31 kernel

                segNzRCIdx.push_back(i-rowStart); 
                segNzRCIdx.push_back(j);
                newVals[pos++] = vals[c];
                cOffset[i-rowStart]++;
                atom[i-rowStart]++;
                nnzInSeg++;

                if ( !cols_seg.count(segId) || cols_seg[segId].back()!=j ){
                    cols_seg[segId].push_back(j);
                }
            }
        }
        if ( (j==last_col && nnzInSeg) || (nnz_limit - nnzInSeg)<=dif || nnzInSeg>nnz_limit ){
        
            // for kernel v31
            if ( !seg_rowPtr.empty() ) seg_rowPtr.push_back( seg_rowPtr.back() + 0 );
            else seg_rowPtr.push_back( 0 );
            
            for ( int i=0; i<tm; ++i ){
                seg_rowPtr.push_back( seg_rowPtr.back() + segcv[i].size() );
                for ( auto &p:segcv[i] ){
                    segNzCV.push_back((float)p.first); // col of the nz is stored in float
                    segNzCV.push_back(p.second);
                }
                segcv[i].clear();
            }


            aux_seg.push({ ridx, segPtr.size()-1 }); // {seg_row, seg_idx}
            id2r[segPtr.size()-1] = ridx;
            count_segs[ridx]++;
            segPtr.push_back(segPtr.back()+nnzInSeg);
            nnzInSeg = 0;
           
            for (int i=rowStart; i<rowStart+tm; ++i){
                if ( i<rowEnd ){
                    if ( atom[i-rowStart]>=0 && atom[i-rowStart]<(rowPtr[i+1]-rowPtr[i]) ){
                        // if the #nz in a specific row of a seg 
                        // is less than that of the whole row,
                        // the row requires "atomic add".
                        // use MSB to mark it.
                        segVoMap.push_back( voMp[i] | (1<<31) );
                    }else{ 
                        segVoMap.push_back( voMp[i] );
                    }
                }else{
                    // for the last panel, the rows may be less than tm 
                    segVoMap.push_back(1<<(bit_width((uint)m)+1));
                }
                
                atom[ i-rowStart ] = 0;
            }
        }
    }
}

void
Mat::stats_collect2(FILE *stream)
{
  //const uint seg_m = ( m + tm - 1 ) / tm;
  //const uint seg_m_floor = m / tm;
  
  const uint seg_nnz_lim = tm * n;
  assert( seg_nnz_lim == tm * uint64_t(n) ); // Overflow check.
  const uint seg_lg_nnz_lim = bit_width(seg_nnz_lim);
  uint seg_lg_nnz_max = 0, seg_lg_nnz_min = seg_lg_nnz_lim;
  seg_lg_nnz_histo.resize(seg_lg_nnz_lim+1);

  const uint n_segs = segPtr.size()-1;

  n_col_sum = 0;
  int sp_seg1 = 0;
  int sp_seg2 = 0;
  int sp_seg3 = 0;
  int sp_seg4 = 0;
  for ( uint seg_idx = 0; seg_idx < n_segs; seg_idx++ )
    {
      const uint nnz_seg = segPtr[seg_idx+1] - segPtr[seg_idx];
      if ( nnz_seg<=NNZ_LIMIT/4 ) sp_seg1++;
      else if ( nnz_seg<=NNZ_LIMIT/2 ) sp_seg2++;
      else if ( nnz_seg<=NNZ_LIMIT ) sp_seg3++;
      else if ( nnz_seg>NNZ_LIMIT ) sp_seg4++;
      
      const uint lg_nnz = bit_width(nnz_seg);
      set_max( seg_lg_nnz_max, lg_nnz );
      set_min( seg_lg_nnz_min, lg_nnz );
      seg_lg_nnz_histo[lg_nnz]++;
      
      unordered_set<int> colset; 
      for (int i=segPtr[seg_idx]; i<segPtr[seg_idx+1]; ++i){
        colset.insert(segNzColIdx[i]);
      }
      n_col_sum += colset.size();
    }

  for ( uint32_t v: segVoMap ) if ( v & (1u<<31) ) atomic_op++;

  if ( !stream ) return;

  fprintf(stream, "Ordering %s.  Segments %d × *\n",
          dl.vertex_order_abbr.c_str(), tm);

  fprintf(stream, "Histogram of lg non-zeros per tile-segment.\n");

  pTable lg_nnz_tab(stream);
  for ( auto i: views::iota(seg_lg_nnz_min,seg_lg_nnz_max+1) )
    {
      pTable_Row _(lg_nnz_tab);
      lg_nnz_tab.entry("Lg", "%2d", i);
      lg_nnz_tab.entry("Seg", "%7d", seg_lg_nnz_histo[i] );
      lg_nnz_tab.entry
        ("Pct", "%6.2f", seg_lg_nnz_histo[i] * 100.0 / n_segs );
    }

  fprintf(stream,"Arrays m=%d, n=%d, k=%d. tile-seg %d × *.\n"
          "              n_segs=%u, nnz=%d  Avg deg=%.1f\n",
         m, n, k, tm, n_segs, nnz, double(nnz)/m);

  fprintf(stream,"nnz / seg: %.3f     Load / B elt  %.3f\n",
         double(nnz) / n_segs,
         n_col_sum / double(nnz) );
  fprintf(stream,"nnz in segs:  (0,%d]: %d  %.3f     (%d,%d]: %d  %.3f     (%d,%d]: %d  %.3f      (%d,%d]: %d  %.3f\n",
         NNZ_LIMIT/4, sp_seg1, double(sp_seg1) / n_segs,
         NNZ_LIMIT/4, NNZ_LIMIT/2, sp_seg2, double(sp_seg2) / n_segs,
         NNZ_LIMIT/2, NNZ_LIMIT, sp_seg3, double(sp_seg3) / n_segs,
         NNZ_LIMIT, NNZ_LIMIT+tm, sp_seg4, double(sp_seg4) / n_segs );
  fprintf(stream,"\n");
}

// convert a row of tiles to FlexSpTiles
void Mat::csr2flex_Rmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

        const int n_tiles_limit = ( n + tn - 1 ) / tn;

	// keep track of the cols in each row
	std::vector<int> cIdx(tm, -1); 
	std::vector<int> cOffset(tm, 0);
	// get the left bound
	// iterate over rows to get the smallest col idx
	unsigned int left = n;
	for (int i=rowStart; i<rowEnd; ++i){
		// here, we assume there is no empty row
		left = min((int)left, (int)colIdx[rowPtr[i]]);
		cIdx[i-rowStart] = colIdx[rowPtr[i]];
	}

	// right bound (exclusive)
	unsigned int right = min((int)left + tn, n);
	int nnzInRows = 0;
    int tiles_in_cur_row = 0;

    //int tileStart = rowPtr[ridx];
    while (pos<rowPtr[rowEnd]){
		int nnzInTile = 0;
        tiles_in_cur_row++;
        assert( tiles_in_cur_row <= n_tiles_limit );
		// collect tiles in the tile-row
        int bit_map = 0;
		for (int i=rowStart; i<rowEnd; ++i){
			// absolute position of the nze in csr, idx = base + offset
			int c = rowPtr[i] + cOffset[i-rowStart];
			//  #nze in the i-th row
			
			// c check is necessary because it constraines nze within the i-th row
            while ( c<rowPtr[i+1] && colIdx[c]<right ){
                //char rc = 0;
                int rc16 = 0;

				// currently, it is not 4-bit
				int temp_rowOffset = i-rowStart;
                //rc |= (temp_rowOffset<<4);
                rc16 |= (temp_rowOffset<<16);

				// real col idx
				int temp_tileColIdx = cIdx[i-rowStart];
                                assert( temp_tileColIdx >= left );
                //rc |= (temp_tileColIdx-left);
                rc16 |= (temp_tileColIdx-left);
			    bit_map |= 1<<(temp_tileColIdx-left);	
                
                // nze values
				newVals[pos] = vals[c];
                rcOffset.push_back(rc16);

				cIdx[i-rowStart] = colIdx[++c];
				pos++;
				cOffset[i-rowStart]++;
				nnzInTile++;
				nnzInRows++;
			}
		}
        
        // ---------- v4 -------
        //tileStart = tileNnz.back()+nnzInTile; 
        // mark the last tile in current row-tile
        //if (pos>=rowPtr[rowEnd]){
        //    nnzInTile |= (1<<31);
        //}
        nnzTile.push_back(nnzInTile); 
        bitMap.push_back(bit_map); 
        // ---------------------
		
		tileNnz.push_back(tileNnz.back()+nnzInTile);
        tileColIdx.push_back(left);
        // update left and right bound for next tile
		left = n;
		for (int i=rowStart; i<rowEnd; ++i){
			// check whether the column goes to the next row
			int rnnz = rowPtr[i+1]-rowPtr[i];
			if (cOffset[i-rowStart]<rnnz){
				left = min((int)left, (int)cIdx[i-rowStart]);
			}
		}
		right = min((int)left + tn, n);

	}
	tileRowPtr.push_back(tileRowPtr.back()+tiles_in_cur_row);
}

// convert a row of tiles to FlexSpTiles
void Mat::csr2flex_Cmajor(int ridx){
	// row tile upper bound and lower bound
	int rowStart = ridx * tm;
	int rowEnd = min(m, (ridx+1)*tm); // exclusive

    const int n_tiles_limit = ( n + tn - 1 ) / tn;

	// keep track of the cols in each row
	std::vector<int> cIdx(tm, -1); 
	std::vector<int> cOffset(tm, 0);
	// get the left bound
	// iterate over rows to get the smallest col idx
	unsigned int left = n;
	for (int i=rowStart; i<rowEnd; ++i){
		// here, we assume there is no empty row
		left = min((int)left, (int)colIdx[rowPtr[i]]);
		cIdx[i-rowStart] = colIdx[rowPtr[i]];
	}

	// right bound (exclusive)
	unsigned int right = min((int)left + tn, n);
	int nnzInRows = 0;
    int tiles_in_cur_row = 0;

    //int tileStart = rowPtr[ridx];
    while (pos<rowPtr[rowEnd]){
		int nnzInTile = 0;
        tiles_in_cur_row++;
        assert( tiles_in_cur_row <= n_tiles_limit );
		// collect tiles in the tile-row
        int bit_map = 0;
        for (int i_tn=0; i_tn<tn; ++i_tn){
            
		    for (int i=rowStart; i<rowEnd; ++i){
                // absolute position of the nze in csr, idx = base + offset
                int c = rowPtr[i] + cOffset[i-rowStart];
                if ( colIdx[c]==left+i_tn && c<rowPtr[i+1] && colIdx[c]<right ){
                    int rc16 = 0;

                    // currently, it is not 4-bit
                    int temp_rowOffset = i-rowStart;
                    //rc |= (temp_rowOffset<<4);
                    rc16 |= (temp_rowOffset<<16);

                    // real col idx
                    int temp_tileColIdx = cIdx[i-rowStart];
                    assert( temp_tileColIdx >= left );
                    //rc |= (temp_tileColIdx-left);
                    rc16 |= (temp_tileColIdx-left);
			        bit_map |= 1<<(temp_tileColIdx-left);	
                    
                    // nze values
                    newVals[pos] = vals[c];
                    rcOffset.push_back(rc16);

                    cIdx[i-rowStart] = colIdx[++c];
                    pos++;
                    cOffset[i-rowStart]++;
                    nnzInTile++;
                    nnzInRows++; 
                }
            }
        }
        nnzTile.push_back(nnzInTile); 	
        bitMap.push_back(bit_map); 
		
        tileNnz.push_back(tileNnz.back()+nnzInTile);
        tileColIdx.push_back(left);
        // update left and right bound for next tile
		left = n;
		for (int i=rowStart; i<rowEnd; ++i){
			// check whether the column goes to the next row
			int rnnz = rowPtr[i+1]-rowPtr[i];
			if (cOffset[i-rowStart]<rnnz){
				left = min((int)left, (int)cIdx[i-rowStart]);
			}
		}
		right = min((int)left + tn, n);
	}
	tileRowPtr.push_back(tileRowPtr.back()+tiles_in_cur_row);
}

void
Mat::stats_collect(FILE *stream)
{
  const uint tmn = tm * tn;

  const uint tile_m = ( m + tm - 1 ) / tm;
  const uint tile_m_floor = m / tm;
  const uint tile_n = ( n + tn - 1 ) / tn;
  tile_p_row_histo.resize(tile_n+1);
  uint max_n_tiles = 0;

  const uint panel_nnz_lim = tm * n;
  assert( panel_nnz_lim == tm * uint64_t(n) ); // Overflow check.
  const uint panel_lg_nnz_lim = bit_width(panel_nnz_lim);
  uint panel_lg_nnz_max = 0, panel_lg_nnz_min = panel_lg_nnz_lim;
  panel_lg_nnz_histo.resize(panel_lg_nnz_lim+1);

  for ( uint tile_r = 0;  tile_r < tile_m;  tile_r++ )
    {
      const uint n_tiles = tileRowPtr[tile_r+1] - tileRowPtr[tile_r];
      assert( n_tiles <= tile_n );
      set_max( max_n_tiles, n_tiles );
      tile_p_row_histo[n_tiles]++;
      if ( tile_r >= tile_m_floor ) continue;
      const auto tile_start = tileRowPtr[tile_r];
      const auto tile_stop = tileRowPtr[tile_r+1];
      const auto tidx_start = tileNnz[tile_start];
      const auto tidx_stop = tileNnz[tile_stop];
      if (tidx_stop <= tidx_start){
        printf("tm = %d, tn = %d, tidx_stop = %d, tidx_satrt = %d\n",tm, tn, tidx_stop, tidx_start);
      }
      assert( tidx_stop > tidx_start );
      const uint nnz_panel = tidx_stop - tidx_start;
      const uint lg_nnz = bit_width(nnz_panel);
      assert( lg_nnz < panel_lg_nnz_histo.size() );
      assert( lg_nnz );
      set_max( panel_lg_nnz_max, lg_nnz );
      set_min( panel_lg_nnz_min, lg_nnz );
      panel_lg_nnz_histo[lg_nnz]++;
    }
  tile_p_row_histo.resize(max_n_tiles+1);
  const uint n_tiles = nnzTile.size();

  tile_nnz_histo.resize(tmn+1);
  n_col_sum = 0;
  uint max_t_nnz = 0;
  for ( uint t_idx = 0; t_idx < n_tiles; t_idx++ )
    {
      const auto nnz = nnzTile[t_idx];
      tile_nnz_histo[nnz]++;
      set_max( max_t_nnz, nnz );
      const uint n_col = popcount(uint(bitMap[t_idx]));
      n_col_sum += n_col;
    }
  tile_nnz_histo.resize(max_t_nnz+1);

  vector<int> tiles_bucket(6,0);
  int total = 0;
  int remain = 0;
  for (int i=0; i<tile_p_row_histo.size(); ++i){
    int counts = tile_p_row_histo[i]; 
    if (counts>=1 && counts<8){
        tiles_bucket[0] += counts;
    }else if (counts>=8 && counts<16){
        tiles_bucket[1] += counts;
    }else if (counts>=16 && counts<32){
        tiles_bucket[2] += counts;
    }else if (counts>=32 && counts<64){
        tiles_bucket[3] += counts;
    }else if (counts>=64 && counts<128){
        tiles_bucket[4] += counts;
    }else if (counts>=128){
        tiles_bucket[5] += counts;
    }else{
        remain += counts;
    }
    total += counts;
  }
  if ( !stream ) return;

  fprintf(stream, "Ordering %s.  Tile %d × %d\n",
          dl.vertex_order_abbr.c_str(), tm, tn);

  fprintf(stream, "Histogram of lg non-zeros per panel (tile row).\n");

  pTable lg_nnz_tab(stream);
  for ( auto i: views::iota(panel_lg_nnz_min,panel_lg_nnz_max+1) )
    {
      pTable_Row _(lg_nnz_tab);
      lg_nnz_tab.entry("Lg", "%2d", i);
      lg_nnz_tab.entry("Panels", "%7d", panel_lg_nnz_histo[i] );
      lg_nnz_tab.entry
        ("Pct", "%6.2f", panel_lg_nnz_histo[i] * 100.0 / tile_m_floor );
    }

  fprintf(stream,"[1,8): %f%%   ", tiles_bucket[0]*100.0/tile_m); 
  fprintf(stream,"[8,16): %f%%    ", tiles_bucket[1]*100.0/tile_m); 
  fprintf(stream,"[16,32): %f%%   ", tiles_bucket[2]*100.0/tile_m); 
  fprintf(stream,"[32,64): %f%%   ", tiles_bucket[3]*100.0/tile_m); 
  fprintf(stream,"[64,128): %f%%  ", tiles_bucket[4]*100.0/tile_m); 
  fprintf(stream,"[128, +OO): %f%%\n", tiles_bucket[5]*100.0/tile_m); 

  fprintf(stream,"Arrays m=%d, n=%d, k=%d. Tile %d × %d.   nnz=%d  Avg deg=%.1f\n",
         m, n, k, tm, tn, nnz, double(nnz)/m);

  fprintf(stream,"nnz / tile: %.3f  Load / B elt  %.3f\n",
         double(nnz) / n_tiles,
         n_col_sum / double(nnz) );
  int n_t_hist_pr = 0;
  fprintf(stream,"Tile nnz histogram: (n_tiles %d)\n",n_tiles);
  for ( uint i=0; i<tile_nnz_histo.size(); i++ )
    if ( auto tnnz = tile_nnz_histo[i]; tnnz )
      {
        if ( n_t_hist_pr++ > 6 ) break;
        fprintf(stream,"%3d %5.2f%%, ", i, tnnz * 100.0 / n_tiles);
      }
  fprintf(stream,"\n");
}


void Mat::print2(){
#ifdef DEBUG
    for (int i=0; i<tileRowPtr.size(); ++i)
		std::cout<<tileRowPtr[i]<<" ";
	std::cout<<std::endl;
    
	/*	
    for (int i=0; i<tileLeftColIdx.size(); ++i)
		std::cout<<tileLeftColIdx[i]<<" ";
	std::cout<<std::endl;
    std::cout<<"------- tile elements: -------"<<std::endl;
	std::cout<<std::endl;
	for (int i=0; i<tileColIdx.size(); ++i)
		std::cout<<tileColIdx[i]<<" ";
    std::cout<<std::endl<<"rc:"<<std::endl;
	for (int i=0; i<rc_Offset.size(); ++i)
		std::cout<<(int)rc_Offset[i]<<" ";
	std::cout<<std::endl;
	for (int i=0; i<newVals.size(); ++i)
		std::cout<<newVals[i]<<" ";
	std::cout<<std::endl;
    */
#endif
    std::cout<<std::endl<<"nnzTile:"<<std::endl;
    for (int i=0; i<nnzTile.size(); ++i){
    //for (int i=0; i<20; ++i){
        std::cout<<nnzTile[i]<<" ";
    }
#ifndef COL_MAJ_TILE
    std::cout<<std::endl<<"bitMap:"<<std::endl;
    //for (int i=0; i<bitMap.size(); ++i){
    for (int i=0; i<20; ++i){
        std::cout<<bitMap[i]<<" ";
    }
#endif
    std::cout<<std::endl<<"rc:"<<std::endl;
	for (int i=0; i<rcOffset.size(); ++i){
	//for (int i=0; i<20; ++i){
		int r = rcOffset[i]>>16;
		int c = rcOffset[i] & 0x0000FFFF;
        std::cout<<"{"<<r<<","<<c<<"}"<<" ";
    }
	
//    std::cout<<std::endl<<"vals:"<<std::endl;
//	for (int i=0; i<newVals.size(); ++i)
//		std::cout<<newVals[i]<<" ";
	std::cout<<std::endl;
	std::cout<<"Flex Tiles: "<<tileNnz.size()-1<<std::endl;
}
